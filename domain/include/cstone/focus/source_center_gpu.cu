#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2022 CSCS, ETH Zurich
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief  Compute cell mass centers for use in focus tree
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/primitives/math.hpp"
#include "source_center.hpp"
#include "source_center_gpu.h"

namespace cstone
{

template<class Tc, class Tm, class Tf, class KeyType>
__global__ void computeLeafSourceCenterKernel(const Tc* x,
                                              const Tc* y,
                                              const Tc* z,
                                              const Tm* m,
                                              const TreeNodeIndex* leafToInternal,
                                              TreeNodeIndex numLeaves,
                                              const LocalIndex* layout,
                                              const KeyType* nodeKeys,
                                              const Box<Tc> box,
                                              Vec4<Tf>* centers)
{
    TreeNodeIndex leafIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (leafIdx >= numLeaves) { return; }

    TreeNodeIndex nodeIdx = leafToInternal[leafIdx];
    centers[nodeIdx]      = massCenter<Tf>(x, y, z, m, layout[leafIdx], layout[leafIdx + 1]);

    KeyType nodeKey  = decodePlaceholderBit(nodeKeys[nodeIdx]);
    int prefixLength = decodePrefixLength(nodeKeys[nodeIdx]);

    IBox cellBox              = sfcIBox(sfcKey(nodeKey), prefixLength / 3);
    auto [geoCenter, geoSize] = centerAndSize<KeyType>(cellBox, box);

    if (layout[leafIdx] == layout[leafIdx + 1])
    {
        centers[nodeIdx] = {geoCenter[0], geoCenter[1], geoCenter[2], Tf(0)};
    }

    Vec3<Tc> dX = makeVec3(centers[nodeIdx]) - geoCenter;
    float diagonal = sqrt(norm2(geoSize));
    assert(diagonal >= sqrt(norm2(dX)));

    //if (diagonal < sqrt(norm2(dX)))
    //{
    //    printf("box %d, [%d - %d], diag %f, norm(dX) %f, %f %f %f\n", nodeIdx, layout[leafIdx], layout[leafIdx + 1],
    //           diagonal, sqrt(norm2(dX)), geoCenter[0], geoCenter[1], geoCenter[2]);
    //    for (LocalIndex i = layout[leafIdx]; i < layout[leafIdx + 1]; ++i)
    //    {
    //        Vec3<Tc> part{x[i], y[i], z[i]};
    //        auto d = minDistance(part, geoCenter, geoSize, box);
    //        printf("  %d %f %f %f\n", nodeIdx, x[i], y[i], y[i]);
    //        assert(sqrt(norm2(d)) < 1e-10);
    //    }
    //}
}

template<class Tc, class Tm, class Tf, class KeyType>
void computeLeafSourceCenterGpu(const Tc* x,
                                const Tc* y,
                                const Tc* z,
                                const Tm* m,
                                const TreeNodeIndex* leafToInternal,
                                TreeNodeIndex numLeaves,
                                const LocalIndex* layout,
                                const KeyType* nodeKeys,
                                const Box<Tc>& box,
                                Vec4<Tf>* centers)
{
    unsigned numThreads = 256;
    unsigned numBlocks  = iceil(numLeaves, numThreads);

    computeLeafSourceCenterKernel<<<numBlocks, numThreads>>>(x, y, z, m, leafToInternal, numLeaves, layout, nodeKeys,
                                                             box, centers);
}

#define COMPUTE_LEAF_SOURCE_CENTER_GPU(Tc, Tm, Tf, KeyType)                                                            \
    template void computeLeafSourceCenterGpu(const Tc*, const Tc*, const Tc*, const Tm*, const TreeNodeIndex*,         \
                                             TreeNodeIndex, const LocalIndex*, const KeyType*, const Box<Tc>&,         \
                                             Vec4<Tf>*);

COMPUTE_LEAF_SOURCE_CENTER_GPU(double, double, double, uint32_t);
COMPUTE_LEAF_SOURCE_CENTER_GPU(double, float, double, uint32_t);
COMPUTE_LEAF_SOURCE_CENTER_GPU(float, float, float, uint32_t);
COMPUTE_LEAF_SOURCE_CENTER_GPU(double, double, double, uint64_t);
COMPUTE_LEAF_SOURCE_CENTER_GPU(double, float, double, uint64_t);
COMPUTE_LEAF_SOURCE_CENTER_GPU(float, float, float, uint64_t);

template<class T>
__global__ void upsweepCentersKernel(TreeNodeIndex firstCell,
                                     TreeNodeIndex lastCell,
                                     const TreeNodeIndex* childOffsets,
                                     SourceCenterType<T>* centers)
{
    const int cellIdx = blockIdx.x * blockDim.x + threadIdx.x + firstCell;
    if (cellIdx >= lastCell) return;

    TreeNodeIndex firstChild = childOffsets[cellIdx];

    if (firstChild) { centers[cellIdx] = CombineSourceCenter<T>{}(cellIdx, firstChild, centers); }
}

template<class T>
void upsweepCentersGpu(int numLevels,
                       const TreeNodeIndex* levelRange,
                       const TreeNodeIndex* childOffsets,
                       SourceCenterType<T>* centers)
{
    constexpr int numThreads = 256;

    for (int level = numLevels - 1; level >= 0; level--)
    {
        int numCellsLevel = levelRange[level + 1] - levelRange[level];
        int numBlocks     = (numCellsLevel - 1) / numThreads + 1;
        if (numCellsLevel)
        {
            upsweepCentersKernel<<<numBlocks, numThreads>>>(levelRange[level], levelRange[level + 1], childOffsets,
                                                            centers);
        }
    }
}

template void upsweepCentersGpu(int, const TreeNodeIndex*, const TreeNodeIndex*, SourceCenterType<float>*);
template void upsweepCentersGpu(int, const TreeNodeIndex*, const TreeNodeIndex*, SourceCenterType<double>*);

template<class KeyType, class T>
__global__ void computeGeoCentersKernel(
    const KeyType* prefixes, TreeNodeIndex numNodes, Vec3<T>* centers, Vec3<T>* sizes, const Box<T> box)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numNodes) { return; }

    KeyType prefix                  = prefixes[i];
    KeyType startKey                = decodePlaceholderBit(prefix);
    unsigned level                  = decodePrefixLength(prefix) / 3;
    auto nodeBox                    = sfcIBox(sfcKey(startKey), level);
    util::tie(centers[i], sizes[i]) = centerAndSize<KeyType>(nodeBox, box);
}

template<class KeyType, class T>
void computeGeoCentersGpu(
    const KeyType* prefixes, TreeNodeIndex numNodes, Vec3<T>* centers, Vec3<T>* sizes, const Box<T>& box)
{
    unsigned numThreads = 256;
    unsigned numBlocks  = iceil(numNodes, numThreads);
    computeGeoCentersKernel<<<numBlocks, numThreads>>>(prefixes, numNodes, centers, sizes, box);
}

#define GEO_CENTERS_GPU(KeyType, T)                                                                                    \
    template void computeGeoCentersGpu(const KeyType* prefixes, TreeNodeIndex numNodes, Vec3<T>* centers,              \
                                       Vec3<T>* sizes, const Box<T>& box)
GEO_CENTERS_GPU(uint32_t, float);
GEO_CENTERS_GPU(uint32_t, double);
GEO_CENTERS_GPU(uint64_t, float);
GEO_CENTERS_GPU(uint64_t, double);

template<class KeyType, class T>
__global__ void geoMacSpheresKernel(
    const KeyType* prefixes, TreeNodeIndex numNodes, SourceCenterType<T>* centers, float invTheta, Box<T> box)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numNodes) { return; }
    centers[i] = computeMinMacR2(prefixes[i], invTheta, box);
}

//! @brief set @p centers to geometric node centers with Mac radius l * invTheta
template<class KeyType, class T>
void geoMacSpheresGpu(
    const KeyType* prefixes, TreeNodeIndex numNodes, SourceCenterType<T>* centers, float invTheta, const Box<T>& box)
{
    unsigned numThreads = 256;
    unsigned numBlocks  = iceil(numNodes, numThreads);
    geoMacSpheresKernel<<<numBlocks, numThreads>>>(prefixes, numNodes, centers, invTheta, box);
}

#define GEO_MAC_SPHERES_GPU(KeyType, T)                                                                                \
    template void geoMacSpheresGpu(const KeyType* prefixes, TreeNodeIndex numNodes, SourceCenterType<T>* centers,      \
                                   float invTheta, const Box<T>& box)
GEO_MAC_SPHERES_GPU(uint32_t, float);
GEO_MAC_SPHERES_GPU(uint32_t, double);
GEO_MAC_SPHERES_GPU(uint64_t, float);
GEO_MAC_SPHERES_GPU(uint64_t, double);

template<class KeyType, class T>
__global__ void
setMacKernel(const KeyType* prefixes, TreeNodeIndex numNodes, Vec4<T>* macSpheres, float invTheta, const Box<T> box)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numNodes) { return; }

    Vec4<T> center   = macSpheres[i];
    T mac            = computeVecMacR2(prefixes[i], util::makeVec3(center), invTheta, box);
    macSpheres[i][3] = (center[3] != T(0)) ? mac : T(0);
}

template<class KeyType, class T>
void setMacGpu(const KeyType* prefixes, TreeNodeIndex numNodes, Vec4<T>* macSpheres, float invTheta, const Box<T>& box)
{
    unsigned numThreads = 256;
    unsigned numBlocks  = iceil(numNodes, numThreads);
    setMacKernel<<<numBlocks, numThreads>>>(prefixes, numNodes, macSpheres, invTheta, box);
}

#define SET_MAC_GPU(KeyType, T)                                                                                        \
    template void setMacGpu(const KeyType* prefixes, TreeNodeIndex numNodes, Vec4<T>* macSpheres, float invTheta,      \
                            const Box<T>& box)

SET_MAC_GPU(uint32_t, float);
SET_MAC_GPU(uint64_t, float);
SET_MAC_GPU(uint32_t, double);
SET_MAC_GPU(uint64_t, double);

template<class T>
__global__ void moveCentersKernel(const Vec3<T>* src, TreeNodeIndex numNodes, Vec4<T>* dest)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numNodes) { return; }
    dest[i][0] = src[i][0];
    dest[i][1] = src[i][1];
    dest[i][2] = src[i][2];
    dest[i][3] = 1.0;
}

template<class T>
void moveCenters(const Vec3<T>* src, TreeNodeIndex numNodes, Vec4<T>* dest)
{
    unsigned numThreads = 256;
    unsigned numBlocks  = iceil(numNodes, numThreads);
    moveCentersKernel<<<numBlocks, numThreads>>>(src, numNodes, dest);
}

template void moveCenters(const Vec3<double>*, TreeNodeIndex, Vec4<double>*);
template void moveCenters(const Vec3<float>*, TreeNodeIndex, Vec4<float>*);

} // namespace cstone
